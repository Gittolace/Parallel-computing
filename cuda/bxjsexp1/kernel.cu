#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include "hip/hip_runtime.h"
#include ""
#include "malloc.h"
using namespace std;

struct Matrix
{
	int width;
	int height;
	float* elements;
};


__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

__global__ void matMul(Matrix *A, Matrix *B, Matrix *C)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}

int main()
{

	int dev = 0;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, dev);
	cout << "ʹ��GPU device " << dev << ": " << devProp.name << endl;
	cout << "SM��������" << devProp.multiProcessorCount << endl;
	cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << endl;
	cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << endl;
	cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << endl;
	cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << endl;


	int T = 10;

	while (T>1)
	{
		int width = pow(2,T);
		int height = pow(2,T);
		Matrix *A, *B, *C;
		// �����й��ڴ�
		hipMallocManaged((void**)&A, sizeof(Matrix));
		hipMallocManaged((void**)&B, sizeof(Matrix));
		hipMallocManaged((void**)&C, sizeof(Matrix));
		int nBytes = width * height * sizeof(float);
		hipMallocManaged((void**)&A->elements, nBytes);
		hipMallocManaged((void**)&B->elements, nBytes);
		hipMallocManaged((void**)&C->elements, nBytes);

		// ��ʼ������
		A->height = height;
		A->width = width;
		B->height = height;
		B->width = width;
		C->height = height;
		C->width = width;
		for (int i = 0; i < width * height; ++i)
		{
			A->elements[i] = 1.0;
			B->elements[i] = 2.0;
		}

		// ����kernel��ִ������
		dim3 blockSize(1, 1);
		dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
			(height + blockSize.y - 1) / blockSize.y);
		// ִ��kernel
		matMul << < gridSize, blockSize >> > (A, B, C);


		// ͬ��device ��֤�������ȷ����
		hipDeviceSynchronize();
		// ���ִ�н��
		float maxError = 0.0;
		for (int i = 0; i < width * height; ++i)
			maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
		std::cout << "������: " << maxError << std::endl;
		T--;
	}

	return 0;

}